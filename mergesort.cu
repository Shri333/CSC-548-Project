#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <ctime>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/merge.h>
#include <thrust/sort.h>
#include "common.cuh"

#define THREADS_PER_BLOCK 256
using namespace std;

__device__ void merge(float *input, float *output, int left_offset, int left_size, int right_offset, int right_size)
{
  int i = 0, j = 0, k = left_offset;

  while (i < left_size && j < right_size)
  {
    if (input[left_offset + i] < input[right_offset + j])
    {
      output[k++] = input[left_offset + i++];
    }
    else
    {
      output[k++] = input[right_offset + j++];
    }
  }

  while (i < left_size)
  {
    output[k++] = input[left_offset + i++];
  }

  while (j < right_size)
  {
    output[k++] = input[right_offset + j++];
  }
}

__global__ void merge_kern(float *data, float *result, int n, int width)
{
  int global_id = blockIdx.x * blockDim.x + threadIdx.x;
  int left_offset = global_id * 2 * width;
  if (left_offset >= n)
  {
    return;
  }

  int left_size = min(width, n - left_offset);
  int right_offset = left_offset + width;
  int right_size = min(width, n - right_offset);

  merge(data, result, left_offset, left_size, right_offset, right_size);
}

void mergesort(thrust::device_vector<float> &device_data)
{
  int n = device_data.size();
  thrust::device_vector<float> device_output(n);

  float *raw_data = thrust::raw_pointer_cast(device_data.data());
  float *raw_result = thrust::raw_pointer_cast(device_output.data());

  for (int width = 1; width < n; width *= 2)
  {
    int num_blocks = (n + 2 * width * THREADS_PER_BLOCK - 1) / (2 * width * THREADS_PER_BLOCK);
    merge_kern<<<num_blocks, THREADS_PER_BLOCK>>>(raw_data, raw_result, n, width);
    hipDeviceSynchronize();
    checkCudaError();
    hipMemcpy(raw_data, raw_result, n * sizeof(float), hipMemcpyDeviceToDevice);
    checkCudaError();
  }
}

int main(int argc, char **argv)
{
  if (argc != 2)
  {
    std::cerr << "Usage: samplesort <size>" << std::endl;
    exit(EXIT_FAILURE);
  }

  int size = std::stoi(argv[1]);
  thrust::host_vector<float> host_vec = genVec(size);
  thrust::device_vector<float> device_vec(size);
  thrust::copy(host_vec.begin(), host_vec.end(), device_vec.begin());

  printf("\nUnsorted:\n");
  for (size_t i = 0; i < host_vec.size(); ++i)
  {
    printf("\t%f", host_vec[i]);
  }
  printf("\n");
  cout << "Sorting vector of size " << size << "..." << endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  mergesort(device_vec);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  checkCudaError();

  thrust::copy(device_vec.begin(), device_vec.end(), host_vec.begin());

  printf("\nSorted: \n");
  for (size_t i = 0; i < host_vec.size(); ++i)
  {
    printf("\t%f", host_vec[i]);
  }
  printf("\n");

  float milliseconds;
  hipEventElapsedTime(&milliseconds, start, stop);

  cout << "Time: " << milliseconds << " ms" << endl;

#ifdef DEBUG
  if (!sorted(host_vec))
    cout << "vec is not sorted!" << endl;
#endif
  return 0;
}
