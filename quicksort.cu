#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <ctime>
#include <thrust/scan.h>
#include <sstream>

#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include "common.cuh"
#define THREADS_PER_BLOCK 256
using namespace std;

void usage()
{
  cout << "usage: quicksort [k]" << endl;
  cout << "where 2^k is the size of the vector to generate for sorting" << endl;
  exit(1);
}

/**
 * @brief Device helper function to swap elements in an array
 *
 * @param arr array of floating point values
 * @param index_a the position of the element to swap with the value at index_b
 * @param index_b the position of the element to swap with the value at index_a
 */
__device__ void swap(float *arr, int index_a, int index_b)
{
  float temp = arr[index_a];
  arr[index_a] = arr[index_b];
  arr[index_b] = temp;
}

/**
 * @brief Reorders elements less than pivot to the left of the pivot
 * and elements greater than the pivot to the right for the given partition.
 * The input array values are randomly generated so always choosing
 * the rightmost element is equivalent to taking a random partition
 * @param arr the array to reorder
 * @param left the left most index of the subarray to partition
 * @param right the right most index of the subarray to partition
 * @return __device__
 */
__device__ int device_partition(float *arr, int left, int right)
{
  float pivot = arr[right];
  int i = left - 1;
  for (int k = left; k <= right; ++k)
  {
    if (arr[k] < pivot)
    {
      swap(arr, ++i, k);
    }
  }
  swap(arr, i + 1, right);
  return (i + 1);
}

/**
 * @brief Recursive quicksort implementation
 *
 * @param arr the array to sort
 * @param left the left most index of the array to sort
 * @param right the right most index of the array to sort
 */
__device__ void device_quicksort(float *arr, int left, int right)
{
  if (left < right)
  {
    int pivot_index = device_partition(arr, left, right);
    device_quicksort(arr, left, pivot_index - 1);
    device_quicksort(arr, pivot_index + 1, right);
  }
}

/**
 * @brief Responsible for calling device quicksort function
 *
 * @param arr the array to sort
 * @param size the number of elements in the array
 */
__global__ void quicksort_kernel(float *arr, int size)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx == 0)
  {
    device_quicksort(arr, 0, size - 1);
  }
}

/**
 * @brief Performs quicksort on the given array
 *
 * @param arr the array to sort
 * @param size the number of elements in the array
 */
void quicksort(float *arr, int size)
{
  float *device_data;
  hipMalloc(&device_data, size * sizeof(float));
  hipMemcpy(device_data, arr, size * sizeof(float), hipMemcpyHostToDevice);

  int num_blocks = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  quicksort_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(device_data, size);
  hipDeviceSynchronize();

  hipMemcpy(arr, device_data, size * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(device_data);
}

/**
 * @brief Generates an array of floating point values and performs
 * quicksort on the array.
 *
 * @param argc Two arguments are expects: ./quicksort and #floats to generate and sort
 * @param argv The array of arguments
 * @return int The exit code
 */
int main(int argc, char **argv)
{
  // read k from argv[1] where 2^k is the size of the vector to generate
  istringstream ss(argv[1]);
  unsigned int k;
  if (!(ss >> k) || k > sizeof(size_t) * 8 - 1)
  {
    usage();
  }

  // generate vector
  size_t size = 1 << k;
  thrust::host_vector<float> host_vec = genVec(size);
  thrust::device_vector<float> device_vec(size);
  thrust::copy(host_vec.begin(), host_vec.end(), device_vec.begin());

  cout << "Sorting vector of size " << size << "..." << endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  quicksort(thrust::raw_pointer_cast(device_vec.data()), size);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  checkCudaError();

  thrust::copy(device_vec.begin(), device_vec.end(), host_vec.begin());

  float milliseconds;
  hipEventElapsedTime(&milliseconds, start, stop);

  cout << "Time: " << milliseconds << " ms" << endl;

#ifdef DEBUG
  if (!sorted(host_vec))
    cout << "vect is not sorted!" << endl;
#endif
  return 0;
}
