#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>
#include <sstream>
#include <iostream>
#include "common.cuh"
using namespace std;

void usage()
{
  cout << "usage: quicksort [k]" << endl;
  cout << "where 2^k is the size of the vector to generate for sorting" << endl;
  exit(1);
}

__global__ void partition(int *arr, int *low, int *high, int n, int *out_low, int *out_high, int *new_size)
{
  int pos = blockIdx.x * blockDim.x + threadIdx.x;

  if (pos < n)
  {
    int hi = high[pos];
    int lo = low[pos];
    int pivot = arr[hi];
    int i = (lo - 1);
    int temp;
    for (int j = lo; j <= hi - 1; j++)
    {
      if (arr[j] <= pivot)
      {
        i++;
        temp = arr[i];
        arr[i] = arr[j];
        arr[j] = temp;
      }
    }
    temp = arr[i + 1];
    arr[i + 1] = arr[hi];
    arr[hi] = temp;
    int p = (i + 1);

    if (p - 1 > lo)
    {
      int ind = atomicAdd(new_size, 1);
      out_low[ind] = lo;
      out_high[ind] = p - 1;
    }
    if (p + 1 < hi)
    {
      int ind = atomicAdd(new_size, 1);
      out_low[ind] = p + 1;
      out_high[ind] = hi;
    }
  }
}

void quicksort(int arr[], int l, int h)
{
  int *low_indices = (int *)malloc((h - l + 1) * sizeof(int));
  int *high_indices = (int *)malloc((h - l + 1) * sizeof(int));

  int top = -1, *device_data, *dev_low_indices, *dev_high_indices;

  low_indices[++top] = l;
  high_indices[top] = h;

  hipMalloc(&device_data, (h - l + 1) * sizeof(int));
  hipMemcpy(device_data, arr, (h - l + 1) * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc(&dev_low_indices, (h - l + 1) * sizeof(int));
  hipMemcpy(dev_low_indices, low_indices, (h - l + 1) * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc(&dev_high_indices, (h - l + 1) * sizeof(int));
  hipMemcpy(dev_high_indices, high_indices, (h - l + 1) * sizeof(int), hipMemcpyHostToDevice);

  int num_threads = 1;
  int num_blocks = 1;
  int num_subarrays = 1;

  int *device_lows, *device_highs, *device_size;
  hipMalloc(&device_lows, (h - l + 1) * sizeof(int));
  hipMalloc(&device_highs, (h - l + 1) * sizeof(int));
  hipMalloc(&device_size, sizeof(int));

  while (num_subarrays > 0)
  {
    int new_size = 0;
    hipMemcpy(device_size, &new_size, sizeof(int), hipMemcpyHostToDevice);
    partition<<<num_blocks, num_threads>>>(device_data, dev_low_indices, dev_high_indices, num_subarrays, device_lows, device_highs, device_size);

    hipMemcpy(&new_size, device_size, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(dev_low_indices, device_lows, new_size * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(dev_high_indices, device_highs, new_size * sizeof(int), hipMemcpyDeviceToDevice);

    if (new_size < NUM_THREADS)
    {
      num_threads = new_size;
    }
    else
    {
      num_threads = NUM_THREADS;
      num_blocks = new_size / num_threads + (new_size % num_threads == 0 ? 0 : 1);
    }
    num_subarrays = new_size;
    hipMemcpy(arr, device_data, (h - l + 1) * sizeof(int), hipMemcpyDeviceToHost);
  }

  hipFree(device_data);
  hipFree(dev_low_indices);
  hipFree(dev_high_indices);
  hipFree(device_lows);
  hipFree(device_highs);
  hipFree(device_size);
  free(low_indices);
  free(high_indices);
}

int main(int argc, char **argv)
{
  istringstream ss(argv[1]);
  unsigned int k;
  if (!(ss >> k) || k > sizeof(size_t) * 8 - 1)
  {
    usage();
  }

  size_t n = 1 << k;
  size_t bytes = n * sizeof(int);
  int *in = (int *)malloc(bytes);

  for (size_t i = 0; i < n; i++)
  {
    in[i] = rand();
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  quicksort(in, 0, n - 1);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  checkCudaError();

  float milliseconds;
  hipEventElapsedTime(&milliseconds, start, stop);
  cout << milliseconds << endl;

#ifdef DEBUG
  if (!isSorted(in, n))
    cout << "vect is not sorted!" << endl;
#endif

  free(in);
  return EXIT_SUCCESS;
}
