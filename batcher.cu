#include "hip/hip_runtime.h"
// implementation of Batcher's odd-even mergesort
// based on "Sorting Networks and their Applications" by Batcher
// author: Shrihan Dadi (sdadi2)
#include <iostream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "common.cuh"
using namespace std;

void usage() {
    cout << "usage: batcher [k]" << endl;
    cout << "where 2^k is the size of the vector to generate for sorting" << endl;
    exit(1);
}

// kernel for batcher's sorting network swaps
__global__ void batcherOddEvenSwap(float vec[], size_t size, unsigned int phase, unsigned int step) {
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= size) {
        return;
    }

    // partner calculation algorithm based on https://gist.github.com/Bekbolatov/c8e42f5fcaa36db38402
    size_t partner = idx ^ (1 << (phase - 1));
    if (step > 1) {
        size_t scale = 1 << (phase - step);
        size_t box = 1 << step;
        size_t scaledIdx = idx / scale - (idx / scale / box) * box;
        if (scaledIdx == 0 || scaledIdx == box - 1) {
            return;
        }
        partner = (scaledIdx & 1) == 0 ? idx - scale : idx + scale;
    }

    cmpSwap(vec, idx, partner);
}

int main(int argc, char** argv) {
    if (argc != 2) {
        usage();
    }

    // read k from argv[1] where 2^k is the size of the vector to generate
    istringstream ss(argv[1]);
    unsigned int k;
    if (!(ss >> k) || k > sizeof(size_t) * 8 - 1) {
        usage();
    }

    // generate vector
    size_t size = 1 << k;
    thrust::host_vector<float> vec = genVec(size);

    // sort with normalized bitonic sort
    cout << "Sorting vector of size " << size << "..." << endl;
    thrust::device_vector<float> gpuVec = vec;
    float* gpuVecPtr = thrust::raw_pointer_cast(gpuVec.data());
    size_t numBlocks = size / NUM_THREADS;
    if (size % NUM_THREADS != 0) {
        numBlocks++;
    }

    // time sorting
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (unsigned int phase = 1; phase <= k; phase++) {
        for (unsigned int step = phase; step >= 1; step--) {
            batcherOddEvenSwap<<<numBlocks, NUM_THREADS>>>(gpuVecPtr, size, phase, step);
        }
    }
    hipEventRecord(stop);

    // copy gpuVec back into vec
    vec = gpuVec;

    // get time to sort
    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    // print out time to sort
    cout << "Time: " << milliseconds << " ms" << endl;

#ifdef DEBUG
    if (!sorted(vec)) {
        cout << "vec is not sorted!" << endl;
    }
#endif

    return 0;
}
