#include "hip/hip_runtime.h"
// implementation of deterministic sample sort
// based on "Deterministic Sample Sort For GPUs" by Dehne and Zaboli
// author: Shrihan Dadi (sdadi2)
#include <iostream>
#include <sstream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "common.cuh"
using namespace std;

// sample size for sample sort (should be a power of 2 and an even divisor of NUM_THREADS)
#define SAMPLE_SIZE 64

// log base 2 of sample size
#define LOG_SAMPLE_SIZE 6

void usage() {
    cout << "usage: dsample [k]" << endl;
    cout << "where 2^k is the size of the vector to generate for sorting" << endl;
    exit(1);
}

// kernel for sorting with bitonic sort
__global__ void bitonicSort(float vec[], size_t size, unsigned int phase, unsigned int step) {
    size_t idx = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    bitonicSwap(vec, size, phase, step, idx);
}

// sort a partition of the global vector on each block using bitonic sort
__global__ void localBitonicSort(float vec[], size_t size) {
    size_t idx = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;

    // copy partition of vector into shared memory
    __shared__ float sharedVec[NUM_THREADS];
    sharedVec[threadIdx.x] = vec[idx];
    __syncthreads();

    // bitonic sort this sub-vector/partition
    for (unsigned int phase = 1; phase <= LOG_NUM_THREADS; phase++) {
        for (unsigned int step = phase; step >= 1; step--) {
            if (threadIdx.x < NUM_THREADS / 2) {
                bitonicSwap(sharedVec, NUM_THREADS, phase, step, threadIdx.x);
            }
            __syncthreads();
        }
    }

    // copy sorted sub-vector back into vec
    vec[idx] = sharedVec[threadIdx.x];
}

// sample equidistant values from vec into samples
__global__ void sample(float vec[], float samples[]) {
    size_t idx = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    if (idx % SAMPLE_SIZE == 0) {
        samples[idx / SAMPLE_SIZE] = vec[idx];
    }
}

// searches for the index in vec where the given num should be located
__device__ size_t binarySearch(float vec[], size_t size, float num) {
    size_t left = 0, right = size - 1;
    while (left < right) {
        size_t mid = left + (right - left) / 2; // to avoid overflow
        if (vec[mid] < num) {
            left = mid + 1;
        } else {
            right = mid;
        }
    }
    return left;
}

// calculates bucket sizes by indexing each global sample in each subvector
__global__ void calcBucketSizes(float vec[], float globalSamples[], size_t bucketSizes[]) {
    size_t idx = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;

    // copy subvec into shared memory
    __shared__ float sharedVec[NUM_THREADS];
    __shared__ size_t sampleIndices[SAMPLE_SIZE];
    sharedVec[threadIdx.x] = vec[idx];
    __syncthreads();

    // binary search indices of each global sample in subvec
    if (threadIdx.x < SAMPLE_SIZE) {
        size_t sampleIdx = binarySearch(sharedVec, NUM_THREADS, globalSamples[threadIdx.x]);
        sampleIndices[threadIdx.x] = sampleIdx;
    }
    __syncthreads();

    // calculate size of each bucket based on sampleIndices
    // if (threadIdx.x < SAMPLE_SIZE) {
    //     if (threadIdx.x == SAMPLE_SIZE - 1) {
    //         bucketSizes[gridDim.x * threadIdx.x + blockIdx.x] 
    //             = NUM_THREADS - sampleIndices[SAMPLE_SIZE - 1];
    //     } else {
    //         bucketSizes[gridDim.x * threadIdx.x + blockIdx.x] 
    //             = sampleIndices[threadIdx.x] - sampleIndices[threadIdx.x - 1];
    //     }
    // }
}

int main(int argc, char** argv) {
    if (argc != 2) {
        usage();
    }

    // read k from argv[1] where 2^k is the size of the vector to generate
    istringstream ss(argv[1]);
    unsigned int k;
    if (!(ss >> k) || k > sizeof(size_t) * 8 - 1) {
        usage();
    }

    // generate vector
    size_t size = 1 << k;
    thrust::host_vector<float> vec = genVec(size);

    // sort with normalized bitonic sort
    cout << "Sorting vector of size " << size << "..." << endl;
    thrust::device_vector<float> gpuVec = vec;
    float* gpuVecPtr = thrust::raw_pointer_cast(gpuVec.data());

    // time sorting
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    if (size <= NUM_THREADS) {
        // sort the vector with bitonic sort instead when the vector is small enough
        for (unsigned int phase = 1; phase <= k; phase++) {
            for (unsigned int step = phase; step >= 1; step--) {
                bitonicSort<<<1, size / 2>>>(gpuVecPtr, size, phase, step);
            }
        }
    } else {
        // sort a partition of the vector (subvec) on each block w/ bitonic sort
        size_t numBlocks = size / NUM_THREADS;
        localBitonicSort<<<numBlocks, NUM_THREADS>>>(gpuVecPtr, size);

        // sample locally
        size_t localSamplesSize = numBlocks * SAMPLE_SIZE;
        thrust::device_vector<float> localSamples(localSamplesSize);
        float* localSamplesPtr = thrust::raw_pointer_cast(localSamples.data());
        sample<<<numBlocks, NUM_THREADS>>>(gpuVecPtr, localSamplesPtr);

        // sort local samples with bitonic sort
        numBlocks = max((size_t) 1, (localSamplesSize / 2) / NUM_THREADS);
        size_t numThreads = min(localSamplesSize / 2, (size_t) NUM_THREADS);
        for (unsigned int phase = 1; phase <= k - LOG_NUM_THREADS + LOG_SAMPLE_SIZE; phase++) {
            for (unsigned int step = phase; step >= 1; step--) {
                bitonicSort<<<numBlocks, numThreads>>>(localSamplesPtr, localSamplesSize, phase, step);
            }
        }

        // sample globally
        thrust::device_vector<float> globalSamples(SAMPLE_SIZE);
        float* globalSamplesPtr = thrust::raw_pointer_cast(globalSamples.data());
        numBlocks = max((size_t) 1, localSamplesSize / NUM_THREADS);
        numThreads = min(localSamplesSize, (size_t) NUM_THREADS);
        sample<<<numBlocks, numThreads>>>(localSamplesPtr, globalSamplesPtr);

        // calculate sizes of buckets (sample indexing)
        thrust::device_vector<size_t> bucketSizes(localSamplesSize);
        size_t* bucketSizesPtr = thrust::raw_pointer_cast(bucketSizes.data());
        numBlocks = size / NUM_THREADS;
        calcBucketSizes<<<numBlocks, NUM_THREADS>>>(gpuVecPtr, globalSamplesPtr, bucketSizesPtr);
    }
    hipEventRecord(stop);

    // copy gpuVec back into vec
    vec = gpuVec;

    // get time to sort
    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    // print out time to sort
    cout << "Time: " << milliseconds << " ms" << endl;

#ifdef DEBUG
    if (!sorted(vec)) {
        cout << "vec is not sorted!" << endl;
    }
#endif

    return 0;
}
